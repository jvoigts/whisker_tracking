#include "hip/hip_runtime.h"
/* 
 * Author: Alex Krizhevsky (akrizhevsky@google.com)
 */

#include "mat_gpu.h"

/*
 * Block size: 16x16.
 * blockIdx.x determines case in batches of 16*imgsPerThread.
 * blockIdx.y determines 4x4 image region in target image.
 *
 * threadIdx.x determines case.
 * threadIdx.y determines pixel.
 *
 * hidActs:     (numFilters, numModulesY, numModulesX, numImages)
 * filters:     (numColors, filterPixels, numFilters)                               if conv
 *              (numModulesY, numModulesX, numColors, filterPixels, numFilters)     otherwise
 * targets:     (numColors, imgSizeY, imgSizeX, numImages)
 *
 * Each block reconstructs one 4x4 pixels from 16*imgsPerThread cases.
 *
 * Number of filters must be divisible by 16.
 * Number of images must be divisible by 16*imgsPerThread  if checkCaseBounds is false.
 * 16 * imgsPerThread must be divisible by 32.
 *
 * This version loads 32 cases at a time, so it gets full coalescing on that load.
 * It only loads 16 weights at a time, so those aren't fully coalesced.
 * This version conserves shared memory by loading 16 filters at a time rather than 32.
 */
 // 8 1 false false true
template <int imgsPerThread, int numColors, bool scale, bool checkCaseBounds, bool conv>
__global__ void img_acts_color(const float* hidActs, const float* filters, float* targets,
                                   const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                   const int filterSize, const int imgSizeY, const int imgSizeX,
                                   const int paddingStart, const int moduleStride,
                                   const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[numColors*16][16 + 1]; // 16 * 17
    __shared__ float shHidActs[16][16*imgsPerThread]; // 16 * 128

    const int blockCaseIdx = blockIdx.x * 16*imgsPerThread; // 0
    const int numRegionsX = DIVUP(imgSizeX, 4); // 7
    const int blockRegionIdx = blockIdx.y; // 0
    const int blockRegionIdxX = blockRegionIdx % numRegionsX; // 0
    const int blockRegionIdxY = blockRegionIdx / numRegionsX; // 0
    const int blockRegionLeft = blockRegionIdxX * 4; // 0
    const int blockRegionTop = blockRegionIdxY * 4; // 0
    const int pxYInRegion = threadIdx.y / 4; // 0
    const int pxXInRegion = threadIdx.y % 4; // 0
    const int pxY = blockRegionTop + pxYInRegion; // 0, 27
    const int pxX = blockRegionLeft + pxXInRegion; // 0
    const int pxIdx = pxY * imgSizeX + pxX; // 0
    const bool isPxInImg = pxY < imgSizeY && pxX < imgSizeX; // true
    const int numModules = numModulesY * numModulesX; // 576
    const int filterPixels = filterSize * filterSize; // 25
    const int imgPixels = imgSizeX * imgSizeY; // 784
    const int tidx = threadIdx.y * 16 + threadIdx.x; // 0
    const int loadY = tidx / 32; // 0, max 7
    const int loadX = tidx % 32; // 0, max 31

    hidActs += blockCaseIdx + loadY * numImages * numModules + loadX;
    filters += threadIdx.x;
    targets += pxIdx * numImages + blockCaseIdx + threadIdx.x;

    float prod[numColors][imgsPerThread];
    #pragma unroll
    for (int c = 0; c < numColors; c++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[c][i] = 0;
        }
    }
    float* shilterLoad = &shFilters[threadIdx.y][threadIdx.x];
    float* shHidActLoad = &shHidActs[loadY][loadX];    
    
    const int startY = blockRegionTop - paddingStart < filterSize ? 0
                        : 1 + (blockRegionTop - paddingStart - filterSize) / moduleStride;
    const int endY = MIN(numModulesY, 1 + (blockRegionTop + 3 - paddingStart) / moduleStride);
    const int startX = blockRegionLeft - paddingStart < filterSize ? 0
                        : 1 + (blockRegionLeft - paddingStart - filterSize) / moduleStride;
    const int endX = MIN(numModulesX, 1 + (blockRegionLeft + 3 - paddingStart) / moduleStride);

    for (int my = startY; my < endY; my++) { // my = 0, 24        
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInModuleY = pxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) { // mx = 0 
            const int moduleIdx = my * numModulesX + mx; // 0
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInModuleX = pxX - moduleLeft;
 
            const bool isPxInModule = pxInModuleY >= 0 && pxInModuleY < filterSize && pxInModuleX >= 0 && pxInModuleX < filterSize; // true
            const int pxIdxInModule = pxInModuleY * filterSize + pxInModuleX; // 0

            for (int f = 0; f < numFilters; f += 16) { // multiply with 16 filters at a time  // 0
                // Now the threads split up into half-warps, and each half-warp decides if it's interested.
                const float* hLoad = &hidActs[(moduleIdx + f * numModules) * numImages]; // hidActs
                #pragma unroll
                for (int i = 0; i < imgsPerThread * 16; i += 32) { // 0, 32, 64, 96
                    if (!checkCaseBounds || blockCaseIdx + i + loadX < numImages) {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time. // 0, 8
                            shHidActLoad[j * 16 * imgsPerThread + i] = hLoad[j * numModules * numImages + i];
                        }
                    } else {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time.
                            shHidActLoad[j * 16 * imgsPerThread + i] = 0;
                        }
                    }
                }
                
                if (isPxInImg && isPxInModule) {
                    // This half-warp is interested, so it's going to load the weights from this module to its pixel.
                    // Not fully coalesced read :(
                    // But taking out this read entirely only reduces the runtime by ~2.8%, so it isn't costing me much.
                    const float* fLoad = conv ? &filters[pxIdxInModule * numFilters + f]
                                              : &filters[(moduleIdx * numColors * filterPixels + pxIdxInModule) * numFilters + f];
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shilterLoad[c * 16 * (16 + 1)] = fLoad[c * filterPixels * numFilters];
                    }                    
                }

                __syncthreads();
                // Do some actual computation
                if (isPxInImg && isPxInModule) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        #pragma unroll
                        for (int w = 0; w < 16; w++) { // over different filters
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread; i++) {
                                prod[c][i] += shFilters[threadIdx.y + c * 16][w] * shHidActs[w][threadIdx.x + i * 16];                                
                            }
                        }
                    }
                }
                __syncthreads();
            }
        }
    }
    // Not fully coalesced write :(... shmem (and fully coalesced) version is actually slightly slower, though
    if (isPxInImg) {
        if (scale) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleTargets * targets[c * imgPixels * numImages + i * 16] + scaleOutputs * prod[c][i];
                    }
                }
            }
        } else {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleOutputs * prod[c][i];
                    }
                }
            }
        }
    }
}

/*
 * Block size: 16x16.
 * blockIdx.x determines case in batches of 16*imgsPerThread, also color in batches of colorsPerThread.
 *  In essence, blockIdx.x.x = 1..numImages/(16*imgsPerThread)
 *              blockIdx.x.y = 1..numImgColors/colorsPerThread
 * blockIdx.y determines 4x4 image region in target image.
 *
 * threadIdx.x determines case.
 * threadIdx.y determines pixel.
 *
 * hidActs:     (numFilters, numModulesY, numModulesX, numImages)
 * filters:     (numFilterColors, filterPixels, numFilters)                             if conv
 *              (numModulesY, numModulesX, numFilterColors, filterPixels, numFilters)   otherwise
 * targets:     (numImageColors, imgSizeY, imgSizeX, numImages)
 *
 * Each block reconstructs one 4x4 pixels from 16*imgsPerThread cases.
 *
 * numImages must be divisible by 16*imgsPerThread if checkCaseBounds is false.
 * 16 * imgsPerThread must be divisible by 32.
 * numImageColors/numGroups must be divisible by colorsPerThread.
 *
 * This version loads 32 cases at a time, so it gets full coalescing on that load.
 * It only loads 16 weights at a time, so those aren't fully coalesced.
 * This version conserves shared memory by loading 16 filters at a time rather than 32.
 * 
 * To be used when there are 4-16 color channels.
 */
template <int imgsPerThread, int colorsPerThread,  bool scale, bool checkCaseBounds, bool conv>
__global__ void img_acts_mediumcolor(const float* hidActs, const float* filters, float* targets,
                                       const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                       const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart,
                                       const int moduleStride, const int numImgColors, const int numGroups,
                                       const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*16][16 + 1];
    __shared__ float shHidActs[16][16*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,16*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * 16*imgsPerThread;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;
    
    const int numRegionsX = DIVUP(imgSizeX, 4);
    const int blockRegionIdx = blockIdx.y;
    const int blockRegionIdxX = blockRegionIdx % numRegionsX;
    const int blockRegionIdxY = blockRegionIdx / numRegionsX;
    const int blockRegionLeft = blockRegionIdxX * 4;
    const int blockRegionTop = blockRegionIdxY * 4;
    const int pxYInRegion = threadIdx.y / 4, pxXInRegion = threadIdx.y % 4;
    const int pxY = blockRegionTop + pxYInRegion;
    const int pxX = blockRegionLeft + pxXInRegion;
    const int pxIdx = pxY * imgSizeX + pxX;
    const bool isPxInImg = pxY < imgSizeY && pxX < imgSizeX;
    const int numModules = numModulesY * numModulesX;
    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * 16 + threadIdx.x;
    const int loadY = tidx / 32, loadX = tidx % 32;

    hidActs += blockCaseIdx + (blockFilterIdx + loadY) * numImages * numModules + loadX;
    filters += blockFilterIdx + filterColorIdx * filterPixels * numFilters + threadIdx.x;
    targets += imgColorIdx * imgPixels * numImages + pxIdx * numImages + blockCaseIdx + threadIdx.x;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int c = 0; c < colorsPerThread; c++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[c][i] = 0;
        }
    }
    const int startY = blockRegionTop - paddingStart < filterSize ? 0
                        : 1 + (blockRegionTop - paddingStart - filterSize) / moduleStride;
    const int endY = MIN(numModulesY, 1 + (blockRegionTop + 3 - paddingStart) / moduleStride);
    const int startX = blockRegionLeft - paddingStart < filterSize ? 0
                        : 1 + (blockRegionLeft - paddingStart - filterSize) / moduleStride;
    const int endX = MIN(numModulesX, 1 + (blockRegionLeft + 3 - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[threadIdx.y][threadIdx.x];
    float* shHidActLoad = &shHidActs[loadY][loadX];

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInModuleY = pxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            const int moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInModuleX = pxX - moduleLeft;

            const bool isPxInModule = pxInModuleY >= 0 && pxInModuleY < filterSize && pxInModuleX >= 0 && pxInModuleX < filterSize;
            const int pxIdxInModule = pxInModuleY * filterSize + pxInModuleX;

            for (int f = 0; f < numFiltersPerGroup; f += 16) { // multipply with 16 filters at a time
                // Now the threads split up into half-warps, and each half-warp decides if it's interested.
                const float* hLoad = &hidActs[(moduleIdx + f * numModules) * numImages];
                #pragma unroll
                for (int i = 0; i < imgsPerThread * 16; i += 32) {
                    if (!checkCaseBounds || blockCaseIdx + loadX + i < numImages) {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time.
                            shHidActLoad[j * 16 * imgsPerThread + i] = hLoad[j * numModules * numImages + i];
                        }
                    } else {
                        #pragma unroll
                        for (int j = 0; j < 16; j += 8) { // load 16 rows of imgsPerThread*16 cols, 8 * 32 elements at a time.
                            shHidActLoad[j * 16 * imgsPerThread + i] = 0;
                        }
                    }
                }

                if (isPxInImg && isPxInModule) {
                    // This half-warp is interested, so it's going to load the weights from this module to its pixel.
         
                    // Not fully coalesced read :(
                    // But taking out this read entirely only reduces the runtime by ~2.8%, so it isn't costing me much.
                    const float* fLoad = conv ? &filters[pxIdxInModule * numFilters + f]
                                              : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInModule * numFilters + f];
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        shFilterLoad[c * 16 * (16 + 1)] = fLoad[c * filterPixels * numFilters];
                    }
                }

                __syncthreads();
                // Do some actual computation
                if (isPxInImg && isPxInModule) {
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        #pragma unroll
                        for (int w = 0; w < 16; w++) {
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread; i++) {
                                prod[c][i] += shFilters[threadIdx.y + c * 16][w] * shHidActs[w][threadIdx.x + i * 16];
                            }
                        }
                    }
                }
                __syncthreads();
            }
        }
    }
    // Not fully coalesced write :(... shmem (and fully coalesced) version is actually slightly slower, though
    if (isPxInImg) {
        if (scale) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleTargets * targets[c * imgPixels * numImages + i * 16] + scaleOutputs * prod[c][i];
                    }
                }
            }
        } else {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * 16 < numImages) {
                    #pragma unroll
                    for (int c = 0; c < colorsPerThread; c++) {
                        targets[c * imgPixels * numImages + i * 16] = scaleOutputs * prod[c][i];
                    }
                }
            }
        }
    }
}

/*
 * Block size: B_YxB_X.
 * blockIdx.x determines case in batches of B_X*imgsPerThread, also color in batches of B_Y*colorsPerThread.
 *  In essence, blockIdx.x.x = 1..numImages/(B_X*imgsPerThread)
 *              blockIdx.x.y = 1..numImgColors/(B_Y*colorsPerThread)
 * blockIdx.y determines image pixel in target image.
 *
 * threadIdx.x determines case.
 * threadIdx.y determines color.
 *
 * hidActs:     (numFilters, numModulesY, numModulesX, numImages)
 * filters:     (numFilterColors, filterPixels, numFilters)                             if conv
 *              (numModulesY, numModulesX, numFilterColors, filterPixels, numFilters)   otherwise
 * targets:     (numImageColors, imgSizeY, imgSizeX, numImages)
 *
 * Each block reconstructs one B_Y*colorsPerThread colors from 1 pixel from B_X*imgsPerThread cases.
 *
 * numImages must be divisible by B_X*imgsPerThread if checkCaseBounds is false.
 * numFiltersPerGroup must be divisible by filterCacheF.
 * 
 * numFilterColors must be divisible by B_Y*colorsPerThread.
 * B_X*B_Y must be divisible by filterCacheF
 * filterCacheF must be divisible by filterCacheH
 *  
 * This version loads 32 cases at a time, so it gets full coalescing on that load.
 * It only loads filterCacheF weights at a time, so those aren't fully coalesced (depending on size of filterCacheF).
 * 
 * To be used when there are >= 16 color channels.
 */
template <int B_Y, int B_X, int imgsPerThread, int colorsPerThread, int filterCacheF, int filterCacheH, bool scale, bool checkCaseBounds, bool conv>
__global__ void conv_img_acts_manycolor_kepler(const float* hidActs, const float* filters, float* targets,
                                          const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                          const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart, const int moduleStride,
                                          const int numImgColors, const int numGroups,
                                          const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*B_Y][filterCacheF]; 
    __shared__ float shHidActs[filterCacheH][B_X*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * B_X*imgsPerThread;
    
    const int imgColorIdx = (blockIdx.x / numImgBlocks) * B_Y*colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int blockPixelIdx = blockIdx.y;
    const int blockPixelIdxX = blockPixelIdx % imgSizeX;
    const int blockPixelIdxY = blockPixelIdx / imgSizeX;

    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * B_X + threadIdx.x;
    const int hidActLoadY = threadIdx.y, hidActLoadX = threadIdx.x;
    //const int hidActLoadY = tidx / (B_X*imgsPerThread), hidActLoadX = tidx % (B_X*imgsPerThread);
    const int filtersLoadY = tidx / filterCacheF, filtersLoadX = tidx % filterCacheF;
    // nvcc is behaving idiotically again, these useless declarations save registers
    //const int outputY = threadIdx.y, outputX = threadIdx.x; 
    //const int ty = threadIdx.y, tx = threadIdx.x;
    const int numModules = numModulesY * numModulesX;

    hidActs += blockCaseIdx + (blockFilterIdx + hidActLoadY) * numImages * numModules + hidActLoadX;
    filters += blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
    targets += (imgColorIdx + threadIdx.y) * imgPixels * numImages + blockPixelIdx * numImages + blockCaseIdx + threadIdx.x;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int c = 0; c < colorsPerThread; c++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[c][i] = 0;
        }
    }

    const int startY = blockPixelIdxY - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxY - paddingStart - filterSize) / moduleStride;
    const int endY = min(numModulesY, 1 + (blockPixelIdxY - paddingStart) / moduleStride);
    const int startX = blockPixelIdxX - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxX - paddingStart - filterSize) / moduleStride;
    const int endX = min(numModulesX, 1 + (blockPixelIdxX - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[filtersLoadY][filtersLoadX];
    float* shHidActLoad = &shHidActs[hidActLoadY][hidActLoadX];
    //const bool noFLoop = filterCacheF == filterCacheH;
    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInFilterY = blockPixelIdxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            const int moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInFilterX = blockPixelIdxX - moduleLeft;
            
            const int pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX;

            for (int f = 0; f < numFiltersPerGroup; f += filterCacheF) { // multiply with filterCacheF filters at a time
                const float* fLoad = conv ? &filters[pxIdxInFilter * numFilters + f]
                                          : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + f];
                #pragma unroll
                for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
                    if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
                        shFilterLoad[i * filterCacheF] = fLoad[i * filterPixels * numFilters];
                    }
                }
                //#pragma unroll
                
                for (int fh = f; fh < f + filterCacheF; fh += filterCacheH) {
                    //conv_img_acts_manycolor_dummy_fhLoop<B_Y, B_X, imgsPerThread, colorsPerThread, filterCacheF, filterCacheH, checkCaseBounds>(hidActs, shHidActLoad, shHidActs, shFilters, moduleIdx, numImages, hidActLoadY, hidActLoadX, blockCaseIdx, numModules, f, fh, prod);
                    
                    const float* hLoad = &hidActs[(moduleIdx + fh * numModules) * numImages];

                    #pragma unroll
                    for (int j = 0; j < filterCacheH; j += B_Y) {
                        if (filterCacheH % B_Y == 0 || hidActLoadY + j < filterCacheH) {
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread*B_X; i += B_X) {
                                if (!checkCaseBounds || blockCaseIdx + hidActLoadX + i < numImages) {
                                    shHidActLoad[j * B_X * imgsPerThread + i] = hLoad[j * numModules * numImages + i];
                                } else {
                                    shHidActLoad[j * B_X * imgsPerThread + i] = 0;
                                }
                            }
                        }
                    }
                   
                    __syncthreads();
                    
                    // Do some actual computation
                    // Using these variables causes register usage to go from 161 --> 123.
                    // But nonetheless, the high-register version is faster.
                    //const float* shF = &shFilters[threadIdx.y][fh-f];
                    //const float* const shF2 = &shFilters[threadIdx.y][fh];
                    //const float*  shH = &shHidActs[0][threadIdx.x];
                    #pragma unroll
                    for (int w = 0; w < filterCacheH; w++) {
                        #pragma unroll
                        for (int c = 0; c < colorsPerThread; c++) {
                            #pragma unroll
                            for (int i = 0; i < imgsPerThread; i++) {
                                prod[c][i] += shFilters[c * B_Y + threadIdx.y][fh-f + w] * shHidActs[w][threadIdx.x + i * B_X];

                            }
                        }
                    }
                    __syncthreads();
                    
                }
            }
        }
    }
    if (scale) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * B_X < numImages) {
                #pragma unroll
                for (int c = 0; c < colorsPerThread; c++) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * targets[c * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[c][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || blockCaseIdx + threadIdx.x + i * B_X < numImages) {
                #pragma unroll
                for (int c = 0; c < colorsPerThread; c++) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleOutputs * prod[c][i];
                }
            }
        }
    }
}

/*
 * New Titan-optimized stuff.
 */

__device__ __forceinline__ void conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(const int my, const int mx, const int numModulesX,
        const int paddingStart, const int moduleStride, const int blockPixelIdxY, const int blockPixelIdxX, const int filterSize, int &moduleIdx, int &pxIdxInFilter) {
    const int moduleTop = paddingStart + my * moduleStride;
    const int pxInFilterY = blockPixelIdxY - moduleTop;

    moduleIdx = my * numModulesX + mx; // out
    const int moduleLeft = paddingStart + mx * moduleStride;
    const int pxInFilterX = blockPixelIdxX - moduleLeft;

    pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX; // out
}

#define IA_PRELOAD_LOOP(w,offset) _Pragma("unroll") \
for (int i = 0; i < imgsPerThread; i++) { \
    _Pragma("unroll") \
    for (int c = 0; c < colorsPerThread; c++) { \
        prod[c][i] += shFilters[c * B_Y + threadIdx.y][(w)+(offset)] * shHidActs[w][threadIdx.x * imgsPerThread + i]; \
    } \
} \

/*
 * Same loop as above but inverted.
 */
#define IA_PRELOAD_LOOP2(w,offset) _Pragma("unroll") \
for (int c = 0; c < colorsPerThread; c++) { \
    _Pragma("unroll") \
    for (int i = 0; i < imgsPerThread; i++) { \
        prod[c][i] += shFilters[c * B_Y + threadIdx.y][(w)+(offset)] * shHidActs[w][threadIdx.x * imgsPerThread + i]; \
    } \
} \

#define IA_PRELOAD_LOOP3(i,offset) _Pragma("unroll") \
for (int w = 0; w < filterCacheH; w++) { \
    _Pragma("unroll") \
    for (int c = 0; c < colorsPerThread; c++) { \
        prod[c][i] += shFilters[c * B_Y + threadIdx.y][(w)+(offset)] * shHidActs[w][threadIdx.x * imgsPerThread + i]; \
    } \
} \

#define IA_PRELOAD_W(z) wPreload[z] = fLoad[(z) * B_X*B_Y/filterCacheF * filterPixels * numFilters];
#define IA_PRELOAD_W_TX(z) wPreload[z] = tex1Dfetch<float>(filters, filtersLoadOffset + (z) * B_X*B_Y/filterCacheF * filterPixels * numFilters);
#define IA_PRELOAD_H(y,x) if (!checkCaseBounds || myCaseIdx + (x) * B_X < numImages) { \
    hPreload[y][x] =  hLoad[(y) * B_Y * numModules * numImages + (x) * B_X]; \
}
#define IA_PRELOAD_H_TX(y,x) if (!checkCaseBounds || myCaseIdx + (x) * B_X < numImages) { \
    hPreload[y][x] =  tex1Dfetch<float>(hidActs, hidActsLoadOffset + (y) * B_Y * numModules * numImages + (x) * B_X); \
}

template <int B_Y, int B_X, int imgsPerThread, int colorsPerThread, int filterCacheF, int filterCacheH, bool scale, bool checkCaseBounds, bool conv>
__global__ void
__launch_bounds__(256, 2)   // 256 threads per block, 2 blocks per multiprocessor
                            // These launch bounds ensure 25% occupancy (128 registers used)
                            // as oppposed to 13% (130 registers) achieved by defaults.
conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex(hipTextureObject_t hidActs, hipTextureObject_t filters, float* targets,
                                          const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                          const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart, const int moduleStride,
                                          const int numImgColors, const int numGroups,
                                          const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*B_Y][filterCacheF];
    __shared__ float shHidActs[filterCacheH][B_X*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * B_X*imgsPerThread;
    const int myCaseIdx = blockCaseIdx + threadIdx.x;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * B_Y*colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int blockPixelIdx = blockIdx.y;
    const int blockPixelIdxX = blockPixelIdx % imgSizeX;
    const int blockPixelIdxY = blockPixelIdx / imgSizeX;

    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * B_X + threadIdx.x;
//    const int hidActLoadY = threadIdx.y % B_Y, hidActLoadX = threadIdx.x % B_X;
    //const int hidActLoadY = tidx / (B_X*imgsPerThread), hidActLoadX = tidx % (B_X*imgsPerThread);
    const int filtersLoadY = tidx / filterCacheF, filtersLoadX = tidx % filterCacheF;
    // nvcc is behaving idiotically again, these useless declarations save registers
    //const int outputY = threadIdx.y, outputX = threadIdx.x;
    //const int ty = threadIdx.y, tx = threadIdx.x;
    const int numModules = numModulesY * numModulesX;
    const int hidActsOffset = (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
    const int filtersOffset = blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
//    hidActs += (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
//    filters += blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
    targets += (imgColorIdx + threadIdx.y) * imgPixels * numImages + blockPixelIdx * numImages + myCaseIdx;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int i = 0; i < imgsPerThread; i++) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            prod[c][i] = 0;
        }
    }
    
    

    const int startY = blockPixelIdxY - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxY - paddingStart - filterSize) / moduleStride;
    const int endY = min(numModulesY, 1 + (blockPixelIdxY - paddingStart) / moduleStride);
    const int startX = blockPixelIdxX - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxX - paddingStart - filterSize) / moduleStride;
    const int endX = min(numModulesX, 1 + (blockPixelIdxX - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[filtersLoadY][filtersLoadX];
    float* shHidActLoad = &shHidActs[threadIdx.y][threadIdx.x * imgsPerThread];
    //const bool noFLoop = filterCacheF == filterCacheH;

    /*
     * Initial preload
     */
    float hPreload[filterCacheH/B_Y][imgsPerThread]; // [2][4]
    float wPreload[filterCacheF*colorsPerThread/B_X]; // [8]

    int moduleIdx, pxIdxInFilter;
    conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(startY, startX, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                         blockPixelIdxX, filterSize, moduleIdx, pxIdxInFilter);
//    const float* fLoad = conv ? &filters[pxIdxInFilter * numFilters + 0]
//                              : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + 0];
    int filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters + 0
                                                  : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters);
    #pragma unroll
    for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
        if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
            wPreload[i * filterCacheF/(B_X*B_Y)] = tex1Dfetch<float>(filters, filtersLoadOffset + i * filterPixels * numFilters);
        }
    }

//    const float* hLoad = &hidActs[(moduleIdx + 0 * numModules) * numImages];
    int hidActsLoadOffset = hidActsOffset + (moduleIdx + 0 * numModules) * numImages;
    #pragma unroll
    for (int j = 0; j < filterCacheH; j += B_Y) {
        if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    hPreload[j/B_Y][i] = tex1Dfetch<float>(hidActs, hidActsLoadOffset + j * numModules * numImages + i * B_X);
                }
            }
        }
    }

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInFilterY = blockPixelIdxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInFilterX = blockPixelIdxX - moduleLeft;

            pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX;
            int myNext = my, mxNext = mx, moduleIdxNext, pxIdxInFilterNext;
            const bool lastModule = my == endY - 1 && mx == endX - 1;
            if (!lastModule) {
                mxNext = mx + 1 == endX ? startX : mx + 1;
                myNext = my + (mx + 1 == endX);
            }
            conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(myNext, mxNext, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                                 blockPixelIdxX, filterSize, moduleIdxNext, pxIdxInFilterNext);
            for (int f = 0; f < numFiltersPerGroup; f += filterCacheF) { // multiply with filterCacheF filters at a time
                #pragma unroll
                for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
                    if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
                        shFilterLoad[i * filterCacheF] = wPreload[i * filterCacheF/(B_X*B_Y)];
                    }
                }

                filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters + f + filterCacheF
                                                          : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + f + filterCacheF);
                if (f == numFiltersPerGroup - filterCacheF) {
                    filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilterNext * numFilters
                                                              : moduleIdxNext * numFilterColors * filterPixels * numFilters + pxIdxInFilterNext * numFilters);
                }

                #pragma unroll
                for (int j = 0; j < filterCacheH; j += B_Y) {
                    if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
                        #pragma unroll
                        for (int i = 0; i < imgsPerThread; i++) {
                            // NOTE: bank conflicts here!
                            if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                                shHidActLoad[j * B_X * imgsPerThread + i] = hPreload[j/B_Y][i];
                            }
                        }
                    }
                }

                __syncthreads();

                hidActsLoadOffset = hidActsOffset + (moduleIdx + (f + filterCacheH) * numModules) * numImages;

                #pragma unroll
                for (int z = 0; z < 4; ++z) {
                    IA_PRELOAD_LOOP(z,0);
                    IA_PRELOAD_W_TX(z);
                }

                #pragma unroll
                for (int z = 4; z < 12; ++z) {
                    IA_PRELOAD_LOOP(z,0);
                    IA_PRELOAD_H_TX((z-4)/4,z%4);
                }

                #pragma unroll
                for (int z = 12; z < 16; ++z) {
                    IA_PRELOAD_LOOP(z,0);
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < filterCacheH; j += B_Y) {
                    if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
                        #pragma unroll
                        for (int i = 0; i < imgsPerThread; i++) {
                            if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                                shHidActLoad[j * B_X * imgsPerThread + i] = hPreload[j/B_Y][i];
                            }
                        }
                    }
                }

                __syncthreads();

                hidActsLoadOffset = hidActsOffset + (moduleIdx + (f + filterCacheF) * numModules) * numImages;
                if (f == numFiltersPerGroup - filterCacheF) {
                    hidActsLoadOffset = hidActsOffset + moduleIdxNext * numImages;
                }

                #pragma unroll
                for (int z = 0; z < 4; ++z) {
                    IA_PRELOAD_LOOP(z,filterCacheH);
                    IA_PRELOAD_W_TX(z+4);
                }

                #pragma unroll
                for (int z = 4; z < 12; ++z) {
                    IA_PRELOAD_LOOP(z,filterCacheH);
                    IA_PRELOAD_H_TX((z-4)/4, z%4);
                }

                #pragma unroll
                for (int z = 12; z < 16; ++z) {
                    IA_PRELOAD_LOOP(z,filterCacheH);
                }

                __syncthreads();
            }
        }
    }
    if (scale) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * targets[c * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[c][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleOutputs * prod[c][i];
                }
            }
        }
    }
}


template <int B_Y, int B_X, int imgsPerThread, int colorsPerThread, int filterCacheF, int filterCacheH, bool scale, bool checkCaseBounds, bool conv>
__global__ void
//__launch_bounds__(128, 3)   // 128 threads per block, 3 blocks per multiprocessor
conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16(hipTextureObject_t hidActs, hipTextureObject_t filters, float* targets,
                                          const int numModulesY, const int numModulesX, const int numImages, const int numFilters,
                                          const int filterSize, const int imgSizeY, const int imgSizeX, const int paddingStart, const int moduleStride,
                                          const int numImgColors, const int numGroups,
                                          const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[colorsPerThread*B_Y][filterCacheF];
    __shared__ float shHidActs[filterCacheH][B_X*imgsPerThread];

    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockCaseIdx = (blockIdx.x % numImgBlocks) * B_X*imgsPerThread;
    const int myCaseIdx = blockCaseIdx + threadIdx.x;

    const int imgColorIdx = (blockIdx.x / numImgBlocks) * B_Y*colorsPerThread; // color idx globally
    const int numFilterColors = numImgColors / numGroups;
    const int blockGroupIdx = imgColorIdx / numFilterColors;
    const int filterColorIdx = imgColorIdx % numFilterColors; // color idx within group
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockFilterIdx = blockGroupIdx * numFiltersPerGroup;

    const int blockPixelIdx = blockIdx.y;
    const int blockPixelIdxX = blockPixelIdx % imgSizeX;
    const int blockPixelIdxY = blockPixelIdx / imgSizeX;

    const int filterPixels = filterSize * filterSize;
    const int imgPixels = imgSizeY * imgSizeX;
    const int tidx = threadIdx.y * B_X + threadIdx.x;
//    const int hidActLoadY = threadIdx.y % B_Y, hidActLoadX = threadIdx.x % B_X;
    //const int hidActLoadY = tidx / (B_X*imgsPerThread), hidActLoadX = tidx % (B_X*imgsPerThread);
    const int filtersLoadY = tidx / filterCacheF, filtersLoadX = tidx % filterCacheF;
    // nvcc is behaving idiotically again, these useless declarations save registers
    //const int outputY = threadIdx.y, outputX = threadIdx.x;
    //const int ty = threadIdx.y, tx = threadIdx.x;
    const int numModules = numModulesY * numModulesX;

    const int hidActsOffset = (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
    const int filtersOffset = blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;

//    hidActs += (blockFilterIdx + threadIdx.y) * numImages * numModules + myCaseIdx;
//    filters += blockFilterIdx + (filterColorIdx + filtersLoadY) * filterPixels * numFilters + filtersLoadX;
    targets += (imgColorIdx + threadIdx.y) * imgPixels * numImages + blockPixelIdx * numImages + myCaseIdx;

    float prod[colorsPerThread][imgsPerThread];
    #pragma unroll
    for (int i = 0; i < imgsPerThread; i++) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            prod[c][i] = 0;
        }
    }

    const int startY = blockPixelIdxY - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxY - paddingStart - filterSize) / moduleStride;
    const int endY = min(numModulesY, 1 + (blockPixelIdxY - paddingStart) / moduleStride);
    const int startX = blockPixelIdxX - paddingStart < filterSize ? 0
                        : 1 + (blockPixelIdxX - paddingStart - filterSize) / moduleStride;
    const int endX = min(numModulesX, 1 + (blockPixelIdxX - paddingStart) / moduleStride);

    float* shFilterLoad = &shFilters[filtersLoadY][filtersLoadX];
    float* shHidActLoad = &shHidActs[threadIdx.y][threadIdx.x * imgsPerThread];
    //const bool noFLoop = filterCacheF == filterCacheH;

    /*
     * Initial preload
     */
    float hPreload[filterCacheH/B_Y][imgsPerThread]; // [4][4]
    float wPreload[filterCacheF*colorsPerThread/B_X]; // [6]

    int moduleIdx, pxIdxInFilter;
    conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(startY, startX, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                         blockPixelIdxX, filterSize, moduleIdx, pxIdxInFilter);
//    const float* fLoad = conv ? &filters[pxIdxInFilter * numFilters + 0]
//                              : &filters[moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + 0];
    int filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters
                                                : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters);
    #pragma unroll
    for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
        if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
            wPreload[i * filterCacheF/(B_X*B_Y)] = tex1Dfetch<float>(filters, filtersLoadOffset + i * filterPixels * numFilters);
        }
    }

//    const float* hLoad = &hidActs[moduleIdx * numImages];
    int hidActsLoadOffset = hidActsOffset + moduleIdx * numImages;
    #pragma unroll
    for (int j = 0; j < filterCacheH; j += B_Y) {
        if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    hPreload[j/B_Y][i] = tex1Dfetch<float>(hidActs, hidActsLoadOffset + j * numModules * numImages + i * B_X);
                }
            }
        }
    }

    for (int my = startY; my < endY; my++) {
        const int moduleTop = paddingStart + my * moduleStride;
        const int pxInFilterY = blockPixelIdxY - moduleTop;

        for (int mx = startX; mx < endX; mx++) {
            moduleIdx = my * numModulesX + mx;
            const int moduleLeft = paddingStart + mx * moduleStride;
            const int pxInFilterX = blockPixelIdxX - moduleLeft;

            pxIdxInFilter = pxInFilterY * filterSize + pxInFilterX;
            int myNext = my, mxNext = mx, moduleIdxNext, pxIdxInFilterNext;
            const bool lastModule = my == endY - 1 && mx == endX - 1;
            if (!lastModule) {
                mxNext = mx + 1 == endX ? startX : mx + 1;
                myNext = my + (mx + 1 == endX);
            }
            conv_img_acts_manycolor_preload_ty_8_tx_32_c_8_ff_32_fh_16_setCoords(myNext, mxNext, numModulesX, paddingStart, moduleStride, blockPixelIdxY,
                                                                                 blockPixelIdxX, filterSize, moduleIdxNext, pxIdxInFilterNext);
            for (int f = 0; f < numFiltersPerGroup; f += filterCacheF) { // multiply with filterCacheF filters at a time
                #pragma unroll
                for (int i = 0; i < colorsPerThread*B_Y; i+= B_X*B_Y/filterCacheF) {
                    if ((colorsPerThread*B_Y) % (B_X*B_Y/filterCacheF) == 0 || i + filtersLoadY < colorsPerThread*B_Y) {
                        shFilterLoad[i * filterCacheF] = wPreload[i * filterCacheF/(B_X*B_Y)];
                    }
                }

                filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilter * numFilters + f + filterCacheF
                                                          : moduleIdx * numFilterColors * filterPixels * numFilters + pxIdxInFilter * numFilters + f + filterCacheF);
                if (f == numFiltersPerGroup - filterCacheF) {
                    filtersLoadOffset = filtersOffset + (conv ? pxIdxInFilterNext * numFilters
                                                              : moduleIdxNext * numFilterColors * filterPixels * numFilters + pxIdxInFilterNext * numFilters);
                }

                #pragma unroll
                for (int j = 0; j < filterCacheH; j += B_Y) {
                    if (filterCacheH % B_Y == 0 || threadIdx.y + j < filterCacheH) {
                        #pragma unroll
                        for (int i = 0; i < imgsPerThread; i++) {
                            // NOTE: bank conflicts here!
                            if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                                shHidActLoad[j * B_X * imgsPerThread + i] = hPreload[j/B_Y][i];
                            }
                        }
                    }
                }
                hidActsLoadOffset = hidActsOffset + (moduleIdx + (f + filterCacheF) * numModules) * numImages;
                if (f == numFiltersPerGroup - filterCacheF) {
                    hidActsLoadOffset = hidActsOffset + moduleIdxNext * numImages;
                }

                __syncthreads();

                // It seems that there is no point explicitly interleaving loads
                // and computations because the scheduler does that anyway.

                IA_PRELOAD_LOOP2(0,0);
                IA_PRELOAD_LOOP2(1,0);
                IA_PRELOAD_LOOP2(2,0);
                IA_PRELOAD_LOOP2(3,0);
                IA_PRELOAD_LOOP2(4,0);
                IA_PRELOAD_LOOP2(5,0);
                IA_PRELOAD_LOOP2(6,0);
                IA_PRELOAD_LOOP2(7,0);
                IA_PRELOAD_LOOP2(8,0);
                IA_PRELOAD_LOOP2(9,0);
                IA_PRELOAD_LOOP2(10,0);
                IA_PRELOAD_LOOP2(11,0);
                IA_PRELOAD_LOOP2(12,0);
                IA_PRELOAD_LOOP2(13,0);
                IA_PRELOAD_LOOP2(14,0);
                IA_PRELOAD_LOOP2(15,0);

                IA_PRELOAD_W_TX(0);
                IA_PRELOAD_W_TX(1);
                IA_PRELOAD_W_TX(2);
                IA_PRELOAD_W_TX(3);
                IA_PRELOAD_W_TX(4);
                IA_PRELOAD_W_TX(5);

                IA_PRELOAD_H_TX(0,0);
                IA_PRELOAD_H_TX(0,1);
                IA_PRELOAD_H_TX(0,2);
                IA_PRELOAD_H_TX(0,3);
                IA_PRELOAD_H_TX(1,0);
                IA_PRELOAD_H_TX(1,1);
                IA_PRELOAD_H_TX(1,2);
                IA_PRELOAD_H_TX(1,3);
                IA_PRELOAD_H_TX(2,0);
                IA_PRELOAD_H_TX(2,1);
                IA_PRELOAD_H_TX(2,2);
                IA_PRELOAD_H_TX(2,3);
                IA_PRELOAD_H_TX(3,0);
                IA_PRELOAD_H_TX(3,1);
                IA_PRELOAD_H_TX(3,2);
                IA_PRELOAD_H_TX(3,3);

                __syncthreads();
            }
        }
    }
    if (scale) {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * targets[c * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[c][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int c = 0; c < colorsPerThread; c++) {
            #pragma unroll
            for (int i = 0; i < imgsPerThread; i++) {
                if (!checkCaseBounds || myCaseIdx + i * B_X < numImages) {
                    targets[c * B_Y * imgPixels * numImages + i * B_X] = scaleOutputs * prod[c][i];
                }
            }
        }
    }
}

/*
 * hidActs:         (numFilters, numModules, numImages)
 * filters:         (numFilterColors, filterPixels, numFilters)
 * targets:         (overSample, numImgColors, imgPixels, numImages)
 * 
 * Note: all of these convolution routines are optimized for the case when
 * the number of images (i.e. the minibatch size) is a multiple of 128. 
 * Other batch sizes will work, but but I made no attempt whatsoever
 * to make them work fast. 
 */
void _imgActs(MatGPU& hidActs, MatGPU& filters, MatGPU& targets,
              size_t imgSize1, size_t imgSize2, 
              size_t filtSize, size_t padding, bool conv) {
    
    // targets sizes
    int imgSizeX = (int) imgSize1;
    int imgSizeY = (int) imgSize2;
    int filterSize = (int) filtSize;
    int paddingStart = -(int) padding;  
    
    mexAssert(paddingStart <= 0, "ia9");    
    int numModulesY = imgSizeY - 2 * paddingStart + 1 - filterSize;
    int numModulesX = imgSizeX - 2 * paddingStart + 1 - filterSize;    
    int numModules = numModulesY * numModulesX;
        
    int moduleStride = 1;
    int numGroups = 1;
    float scaleTargets = 0;
    float scaleOutput = 1;    
    
    mexAssert(hidActs.stride_ == 1 && filters.stride_ == 1 && targets.stride_ == 1,
            "In _imgActs one of strides is not 1");  
    
    int numImages = (int) targets.size1_;
    int imgPixels = imgSizeY * imgSizeX;
    mexAssert(targets.size2_ % imgPixels == 0, "ia5");
    int numImgColors = (int) targets.size2_ / imgPixels;    
    mexAssert(numImgColors % numGroups == 0, "ia4");
    mexAssert(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)), "ia1");
    
    int numFilterColors = numImgColors / numGroups;
    mexAssert(numGroups == 1 || numFilterColors % 4 == 0, "ia2");
    int numFilters = (int) filters.size1_;
    mexAssert(numFilters % (16*numGroups) == 0, "Number of outputmaps should be divisible by 16");
    int filterModuleMult = conv ? 1 : numModules;    
    mexAssert(filters.size2_ % (filterModuleMult * numFilterColors) == 0, "ia8");
    int filterPixels = (int) filters.size2_ / (filterModuleMult * numFilterColors);
    mexAssert(filterSize * filterSize == filterPixels, "ia7");
    
    mexAssert(hidActs.size1_ == numImages, "ia14");
    mexAssert(hidActs.size2_ == numFilters * numModules, "ia13");
    
    dim3 blocks;
    dim3 threads;
    int colorsPerThread, imgsPerThread;
    if (numFilterColors % 8 == 0) {
        threads = dim3(32, numFilterColors % 64 == 0 ? 8 : 4);
        colorsPerThread = numFilterColors % 64 == 0 ? 8
                        : numFilterColors % 48 == 0 ? 12
                        : numFilterColors % 32 == 0 ? 8
                        : numFilterColors % 16 == 0 ? 4
                        : 2;
        imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;
        mexAssert(numFilterColors % (threads.y * colorsPerThread) == 0, "ia15");
        
        blocks = dim3(DIVUP(numImages, threads.x*imgsPerThread) * (numImgColors/(threads.y*colorsPerThread)), imgPixels);
        // NOTE: the case when channels % 32 == 0 but channels % 48 != 0 and channels % 64 != 0 has not been optimized!!
    } else if (numFilterColors > 3) {
        // NOTE: THIS CASE HAS NOT BEEN OPTIMIZED FOR KEPLER!!
        imgsPerThread = numImages % 128 == 0 ? 8 : numImages % 64 == 0 ? 4 : 2;
        threads = dim3(16, 16);
        colorsPerThread = numFilterColors % 4 == 0 ? 4 : 2;
        blocks = dim3(DIVUP(numImages,threads.x*imgsPerThread) * (numImgColors / colorsPerThread), DIVUP(imgSizeY,4) * DIVUP(imgSizeX,4));
    } else {
        // NOTE: THIS CASE HAS NOT BEEN OPTIMIZED FOR KEPLER!!
        imgsPerThread = numImages % 128 == 0 ? 8 : numImages % 64 == 0 ? 4 : 2;
        threads = dim3(16, 16);
        blocks = dim3(DIVUP(numImages,threads.x*imgsPerThread), DIVUP(imgSizeY,4) * DIVUP(imgSizeX,4));
    }
    bool checkCaseBounds = numImages % (threads.x * imgsPerThread) != 0;    
    hipStream_t stream = MatGPU::_defaultStream;

    if (checkCaseBounds == false) {
        if (numFilterColors % 8 == 0) {
            if (numFilterColors % 64 == 0) {
                mexAssert(numFilters % (32*numGroups) == 0, "Do not use less than 32 outputmaps after >= 64 outputmaps");
                // this code assumes we hvae 32 filters because it uses filter cache of 32!
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8), 32, 4, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_preloadfh_ty_8_tx_32_c_8_ff_32_fh_16_tex< 8, 32, 4, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.getTextureObject(), filters.getTextureObject(), targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 2, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 8, 32, 2, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 48 == 0) {
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4), 32, 4, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_preloadfh_ty_4_tx_32_c_12_ff_16_fh_16< 4, 32, 4, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.getTextureObject(), filters.getTextureObject(), targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 2, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 32 == 0) {
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 4, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 2, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 16 == 0) {                
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 4, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 2, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 8 == 0) {
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 4, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 4, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 2, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 2, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, false, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
        }
        else if (numFilterColors > 3) {
            if (numFilterColors == 4) {
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 8), 4, false, false, true >, hipFuncCachePreferShared);
                    img_acts_mediumcolor < 8, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 4), 4, false, false, true >, hipFuncCachePreferShared);
                    img_acts_mediumcolor < 4, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, false, true >, hipFuncCachePreferShared);
                    img_acts_mediumcolor < 2, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, false, true >, hipFuncCachePreferShared);
                    img_acts_mediumcolor < 2, 4, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors == 2) {
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 8, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 4, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
        }
        else if (numFilterColors <= 3) {
            if (numFilterColors == 3) {
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 3, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 8, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 3, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 4, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 3, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors == 2) {
                if (numImages % 128 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 8, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 4, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 2, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors == 1) {
                if (numImages % 128 == 0) {
                    //mexPrintMsg("yep");
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 8), 1, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 8, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 64 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 4), 1, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 4, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
                else if (numImages % 16 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, false, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 1, false, false, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
        }
    }
    else if (checkCaseBounds == true) {
        if (numFilterColors % 8 == 0) {
            if (numFilterColors % 64 == 0) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 8), 32, 1, 8, 32, 16, false, true, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 8, 32, 1, 8, 32, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 48 == 0) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 12, 16, 16, false, true, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 12, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 32 == 0) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 8, 32, 16, false, true, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 8, 32, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 16 == 0) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 4, 16, 16, false, true, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 4, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors % 8 == 0) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor_kepler < 4), 32, 1, 2, 16, 16, false, true, true >, hipFuncCachePreferShared);
                    conv_img_acts_manycolor_kepler < 4, 32, 1, 2, 16, 16, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
        }
        else if (numFilterColors > 3) {
            if (numFilterColors == 4) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor < 2), 4, false, true, true >, hipFuncCachePreferShared);
                    img_acts_mediumcolor < 2, 4, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors == 2) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, true, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 2, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
        }
        else if (numFilterColors <= 3) {
            if (numFilterColors == 3) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 3, false, true, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 3, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors == 2) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 2, false, true, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 2, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
            else if (numFilterColors == 1) {
                if (numImages % 1 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color < 2), 1, false, true, true >, hipFuncCachePreferShared);
                    img_acts_color < 2, 1, false, true, true ><<<blocks, threads, 0, stream>>>(hidActs.data_, filters.data_, targets.data_, numModulesY, numModulesX, numImages, numFilters, filterSize, imgSizeY, imgSizeX, paddingStart, moduleStride, scaleTargets, scaleOutput);
                }
            }
        }
    }
    mexAssert(hipGetLastError() == hipSuccess, "imgActs: kernel execution failed");
}

